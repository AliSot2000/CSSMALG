//
// Created by alisot2000 on 05.12.22.
//
#ifndef CSSMALG_FASTFW_CU
#define CSSMALG_FASTFW_CU

#include <iostream>
#include <assert.h>

__global__
void GPUInnerLoops(double *dis, int *next, int k, int V) {
    //calculates unique thread ID in the block
    int t = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x); // Absolute Thread ID using block dimensions

    //calculates unique block ID in the grid
    int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x); // Absolute Block ID using grid dimensions

    //block size (this is redundant though)
    int T= blockDim.x*blockDim.y*blockDim.z;

    //grid size (this is redundant though)
    int B= gridDim.x*gridDim.y*gridDim.z;

    double newDistance;
    /*
    * Each cell in the matrix is assigned to a different thread.
    * Each thread do O(number of assigned cell) computation.
    * Assigned cells of different threads does not overlape with
    * each other. And so no need for synchronization.
    */
    for (int i=b; i<V; i+=B)
        {
            for(int j=t; j<V; j+=T)
            {
                newDistance=dis[i * V + k] + dis[k * V + j];
                next[i * V + j] = next[i * V + k] * (newDistance < dis[i * V + j]) + next[i * V + j] * (newDistance >= dis[i * V + j]);
                dis[i * V + j] = newDistance * (newDistance < dis[i * V + j]) + dis[i * V + j] * (newDistance >= dis[i * V + j]);
            }
        }
    }

void FloydWarshal(double* dis, int* next, int V){
    double* distance;
    int* neighbour;

    // Allocate Memory on GPU
    hipMallocManaged(&distance, V*V*sizeof(double));
    hipMallocManaged(&neighbour, V*V*sizeof(int));

    // Copy prepared Array to
    auto result = hipMemcpy(distance, dis, V * V * sizeof(double), hipMemcpyHostToDevice);
    assert(result == hipSuccess);

    result = hipMemcpy(neighbour, next, V * V * sizeof(int), hipMemcpyHostToDevice);
    assert(result == hipSuccess);

    std::cout << std::endl;
    for (int k = 0; k < V; k++)
    {
        std::cout << "\rk: " << (k + 1) << " of " << V;
        GPUInnerLoops<<<dim3(2,1,1),dim3(1024,1,1)>>>(distance,neighbour,k,V);
        hipDeviceSynchronize();
        /*
        for (int i = 0; i < V; i++){
            for (int j = 0; j < V; j++){
                std::cout << distance[i * V + j] << " ";
            }
            std::cout << std::endl;
        }
         */
    }
    std::cout << std::endl;

    result = hipMemcpy(dis, distance, V * V * sizeof(double), hipMemcpyDeviceToHost);
    assert(result == hipSuccess);
    result = hipMemcpy(next, neighbour, V * V * sizeof(int), hipMemcpyDeviceToHost);
    assert(result == hipSuccess);
}

#endif


