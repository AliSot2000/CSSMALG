//
// Created by alisot2000 on 05.12.22.
//
#ifndef CSSMALG_FASTFW_CU
#define CSSMALG_FASTFW_CU

#include <iostream>
#include <assert.h>
#define CUDA_SCALAR 4

__global__
void GPUInnerLoops(double *dis, int *next, int k, int V) {
    //calculates unique thread ID in the block
    int t = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x); // Absolute Thread ID using block dimensions

    //calculates unique block ID in the grid
    int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x); // Absolute Block ID using grid dimensions

    //block size (this is redundant though)
    int T= blockDim.x*blockDim.y*blockDim.z;

    //grid size (this is redundant though)
    int B= gridDim.x*gridDim.y*gridDim.z;

    double newDistance;
    /*
    * Each cell in the matrix is assigned to a different thread.
    * Each thread do O(number of assigned cell) computation.
    * Assigned cells of different threads does not overlape with
    * each other. And so no need for synchronization.
    */
    for (int i=b; i<V; i+=B)
        {
            for(int j=t; j<V; j+=T)
            {
                newDistance=dis[i * V + k] + dis[k * V + j];
                next[i * V + j] = next[i * V + k] * (newDistance < dis[i * V + j]) + next[i * V + j] * (newDistance >= dis[i * V + j]);
                dis[i * V + j] = newDistance * (newDistance < dis[i * V + j]) + dis[i * V + j] * (newDistance >= dis[i * V + j]);
            }
        }
    }

void FloydWarshal(double* dis, int* next, int V){
    double* distance;
    int* neighbour;

    // Allocate Memory on GPU
    auto alresult = hipMallocManaged(&distance, V*V*sizeof(double));
    assert(alresult == hipSuccess && "Failed to allocate memory on GPU for distance");
    alresult = hipMallocManaged(&neighbour, V*V*sizeof(int));
    assert(alresult == hipSuccess && "Failed to allocate memory on GPU for neighbour");

    // Copy prepared Array to
    auto result = hipMemcpy(distance, dis, V * V * sizeof(double), hipMemcpyHostToDevice);
    assert(result == hipSuccess && "Failed to copy distance array to GPU");

    result = hipMemcpy(neighbour, next, V * V * sizeof(int), hipMemcpyHostToDevice);
    assert(result == hipSuccess && "Failed to copy neighbour array to GPU");

    std::cout << std::endl;
    for (int k = 0; k < V; k++)
    {
#ifdef SLURM_OUTPUT
        std::cout << "k: " << (k + 1) << " of " << V << std::cout;
#else
        std::cout << "\rk: " << (k + 1) << " of " << V << std::flush;
#endif

        GPUInnerLoops<<<dim3(CUDA_SCALAR,1,1),dim3(1024,1,1)>>>(distance,neighbour,k,V);
        alresult = hipGetLastError();
        assert(alresult == hipSuccess && "Failed to launch GPUInnerLoops kernel");
        result = hipDeviceSynchronize();
        assert(result == hipSuccess && "Failed to synchronize GPU");
        /*
        for (int i = 0; i < V; i++){
            for (int j = 0; j < V; j++){
                std::cout << distance[i * V + j] << " ";
            }
            std::cout << std::endl;
        }
         */
    }
    std::cout << std::endl;

    result = hipMemcpy(dis, distance, V * V * sizeof(double), hipMemcpyDeviceToHost);
    assert(result == hipSuccess && "Failed to copy distance array to CPU");
    result = hipMemcpy(next, neighbour, V * V * sizeof(int), hipMemcpyDeviceToHost);
    assert(result == hipSuccess && "Failed to copy neighbour array to CPU");
}

#endif


